
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdint>

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

struct foo {
    int* a;
};

__global__ void kernel(foo* f) {
    printf("f->a : %d\n", *(f->a));
}

int main(int argc, char** argv) {
    foo* f = nullptr;
    int bar = 3;
    hipMalloc((void**)&f, sizeof(foo));
    cudaCheckErrors("alloc");
    hipMalloc((void**)&f->a, sizeof(int));
    cudaCheckErrors("alloc2");

    hipMemcpy((&f->a), &bar, sizeof(int), hipMemcpyHostToDevice);
    cudaCheckErrors("memcpy");
    kernel<<<1,1>>>(f);
    cudaCheckErrors("kernel");
    hipDeviceSynchronize();
}