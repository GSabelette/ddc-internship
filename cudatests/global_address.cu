
#include <hip/hip_runtime.h>
#include <stdio.h>

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

__constant__ __device__ int* ptr;

__global__ void kernel() {
    printf("&ptr : %p\n\n", &ptr);
}

int main() {
    void* dev_addr = nullptr;

    kernel<<<1,1>>>();
    cudaCheckErrors("Kernel");
    hipDeviceSynchronize();
    
    hipGetSymbolAddress((void**)&dev_addr, HIP_SYMBOL(ptr));
    cudaCheckErrors("getsymboladd");
    printf("dev addr : %p", dev_addr);
}